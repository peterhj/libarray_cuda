
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdio.h>

#define BANK_OFFSET(idx) ({ __typeof__ (idx) _idx = idx; ((_idx) + ((_idx) / 32)); })

__global__ void map_print_i32_kernel(
    const int32_t *src, int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    printf("DEBUG: print: [%d] %d\n", i, src[i]);
  }
}

extern "C" void array_cuda_map_print_i32(
    const int32_t *src, int n,
    hipStream_t stream)
{
  map_print_i32_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      src, 32);
}

__global__ void map_print_f32_kernel(
    const float *src, int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    printf("DEBUG: print: [%d] %g\n", i, src[i]);
  }
}

extern "C" void array_cuda_map_print_f32(
    const float *src, int n,
    hipStream_t stream)
{
  map_print_f32_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      src, 32);
}

__global__ void map_set_constant_i32_kernel(
    int32_t *src, int n,
    int32_t c)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    src[i] = c;
  }
}

extern "C" void array_cuda_map_set_constant_i32(
    int32_t *src, int n,
    int32_t c,
    hipStream_t stream)
{
  map_set_constant_i32_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      src, n, c);
}

__global__ void map_set_constant_f32_kernel(
    float *src, int n,
    float c)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    src[i] = c;
  }
}

extern "C" void array_cuda_map_set_constant_f32(
    float *src, int n,
    float c,
    hipStream_t stream)
{
  map_set_constant_f32_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      src, n, c);
}

__global__ void map_cast_u8_to_f32_v(
    const uint32_t *vsrc, int vn,
    float *dst, int n)
{
  int vi = threadIdx.x + blockIdx.x * blockDim.x;
  int i0 = 4 * vi;
  int i1 = i0 + 1;
  int i2 = i0 + 2;
  int i3 = i0 + 3;
  if (vi < vn) {
    uint32_t v = vsrc[vi];
    float x0 = (float)(v & 0xff);
    float x1 = (float)((v >> 8) & 0xff);
    float x2 = (float)((v >> 16) & 0xff);
    float x3 = (float)((v >> 24) & 0xff);
    if (i0 < n) {
      dst[i0] = x0;
    }
    if (i1 < n) {
      dst[i1] = x1;
    }
    if (i2 < n) {
      dst[i2] = x2;
    }
    if (i3 < n) {
      dst[i3] = x3;
    }
  }
}

__global__ void map_cast_u8_to_f32_vs(
    const uint32_t *vsrc, int vn,
    float *dst, int n)
{
  __shared__ float cache[4 * (1024 + 32)];
  int vi = threadIdx.x + blockIdx.x * blockDim.x;
  int i0 = threadIdx.x + 4 * blockIdx.x * blockDim.x;
  int i1 = i0 + 1024;
  int i2 = i0 + 2 * 1024;
  int i3 = i0 + 3 * 1024;
  if (vi < vn) {
    uint32_t v = vsrc[vi];
    cache[BANK_OFFSET(4 * threadIdx.x)]     = (float)(v & 0xff);
    cache[BANK_OFFSET(4 * threadIdx.x + 1)] = (float)((v >> 8) & 0xff);
    cache[BANK_OFFSET(4 * threadIdx.x + 2)] = (float)((v >> 16) & 0xff);
    cache[BANK_OFFSET(4 * threadIdx.x + 3)] = (float)((v >> 24) & 0xff);
    __syncthreads();
    if (i0 < n) {
      dst[i0] = cache[BANK_OFFSET(threadIdx.x)];
    }
    if (i1 < n) {
      dst[i1] = cache[BANK_OFFSET(threadIdx.x + 1024)];
    }
    if (i2 < n) {
      dst[i2] = cache[BANK_OFFSET(threadIdx.x + 2 * 1024)];
    }
    if (i3 < n) {
      dst[i3] = cache[BANK_OFFSET(threadIdx.x + 3 * 1024)];
    }
  }
}

extern "C" void array_cuda_map_cast_u8_to_f32_vec(
    const uint8_t *src, int n,
    float *dst,
    hipStream_t stream)
{
  int vn = (n+3)/4;
  map_cast_u8_to_f32_v<<<(vn+1024-1)/1024, 1024, 0, stream>>>(
      (const uint32_t *)src, vn, dst, n);
}

__global__ void map_cast_u8_to_f32_v_n(
    const uint32_t *vsrc, int vn,
    float *dst, int n)
{
  int vi = threadIdx.x + blockIdx.x * blockDim.x;
  int i0 = 4 * vi;
  int i1 = i0 + 1;
  int i2 = i0 + 2;
  int i3 = i0 + 3;
  if (vi < vn) {
    uint32_t v = vsrc[vi];
    float x0 = (float)(v & 0xff) / 255.0f;
    float x1 = (float)((v >> 8) & 0xff) / 255.0f;
    float x2 = (float)((v >> 16) & 0xff) / 255.0f;
    float x3 = (float)((v >> 24) & 0xff) / 255.0f;
    if (i0 < n) {
      dst[i0] = x0;
    }
    if (i1 < n) {
      dst[i1] = x1;
    }
    if (i2 < n) {
      dst[i2] = x2;
    }
    if (i3 < n) {
      dst[i3] = x3;
    }
  }
}

__global__ void map_cast_u8_to_f32_vs_n(
    const uint32_t *vsrc, int vn,
    float *dst, int n)
{
  __shared__ float cache[4 * (1024 + 32)];
  int vi = threadIdx.x + blockIdx.x * blockDim.x;
  int i0 = threadIdx.x + 4 * blockIdx.x * blockDim.x;
  int i1 = i0 + 1024;
  int i2 = i0 + 2 * 1024;
  int i3 = i0 + 3 * 1024;
  if (vi < vn) {
    uint32_t v = vsrc[vi];
    cache[BANK_OFFSET(4 * threadIdx.x)]     = (float)(v & 0xff) / 255.0f;
    cache[BANK_OFFSET(4 * threadIdx.x + 1)] = (float)((v >> 8) & 0xff) / 255.0f;
    cache[BANK_OFFSET(4 * threadIdx.x + 2)] = (float)((v >> 16) & 0xff) / 255.0f;
    cache[BANK_OFFSET(4 * threadIdx.x + 3)] = (float)((v >> 24) & 0xff) / 255.0f;
    __syncthreads();
    if (i0 < n) {
      dst[i0] = cache[BANK_OFFSET(threadIdx.x)];
    }
    if (i1 < n) {
      dst[i1] = cache[BANK_OFFSET(threadIdx.x + 1024)];
    }
    if (i2 < n) {
      dst[i2] = cache[BANK_OFFSET(threadIdx.x + 2 * 1024)];
    }
    if (i3 < n) {
      dst[i3] = cache[BANK_OFFSET(threadIdx.x + 3 * 1024)];
    }
  }
}

extern "C" void array_cuda_map_cast_u8_to_f32_vec_norm(
    const uint8_t *src, int n,
    float *dst,
    hipStream_t stream)
{
  int vn = (n+3)/4;
  map_cast_u8_to_f32_v_n<<<(vn+1024-1)/1024, 1024, 0, stream>>>(
      (const uint32_t *)src, vn, dst, n);
}

__global__ void map_add_i32_kernel(
    const int32_t *src, int n,
    int32_t *dst)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    dst[i] = dst[i] + src[i];
  }
}

extern "C" void array_cuda_map_add_i32(
    const int32_t *src, int n,
    int32_t *dst,
    hipStream_t stream)
{
  map_add_i32_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      src, n, dst);
}

__global__ void map_add_f32_kernel(
    const float *src, int n,
    float *dst)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    dst[i] = dst[i] + src[i];
  }
}

extern "C" void array_cuda_map_add_f32(
    const float *src, int n,
    float *dst,
    hipStream_t stream)
{
  map_add_f32_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      src, n, dst);
}
